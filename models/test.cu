#include "hip/hip_runtime.h"
/**
 * TEST PROGRAM
 */

#include <iostream>
#include <cstring>
#include <cassert>
#include <string>
#include <fstream>
#include <algorithm>
#include <random>

#include "inference/smc/smc.cuh"
#include "trees/tree_utils.cuh"
#include "utils/math.cuh"
#include "utils/stack.cuh"
#include "dists/delayed.cuh"

#include "trees/cetaceans.cuh"
#include "trees/default_trees.cuh"

#define STEP_SIZE 0.1
#define REPEATS 10


/* Program state */
struct progState_t {
  floating_t f = 0;
};


#define NUM_BBLOCKS 1


INIT_MODEL(progState_t)

/*
 * simCombineDS - required BBLOCK
 */
BBLOCK(test,
{
  normalInverseGamma_t alphaSigma = normalInverseGamma_t(0, 1.0, 3.0, 0.05);
  PSTATE.f = 0;
  for (int i = 0; i < REPEATS; i++) {
    floating_t f =  SAMPLE(linearNormalInverseGammaNormal, alphaSigma, STEP_SIZE, 0, STEP_SIZE);
    PSTATE.f += f;
  }

  NEXT = NULL;
})

CALLBACK(printResults, {
  floating_t meanF = 0;
  floating_t varF = 0;
  for (int i = 0; i < N; i++) {
    meanF += PSTATES[i].f;
  }
  meanF = meanF / N; 
  for (int i = 0; i < N; i++ ) {
    varF += pow(meanF - PSTATES[i].f, 2.0);
  }
  varF = varF / (N - 1);
  printf("mean, var\n");
  printf("%f, %f\n", meanF, varF);
})

CALLBACK(dumpSample, {
    //printf("multiplier\n");
    for (int i = 0; i < N; i++) {
      printf("%f\n", PSTATES[i].f);
    }
  })

MAIN({
    FIRST_BBLOCK(test)
    //SMC(printResults)
    SMC(dumpSample)
    //SMC(NULL)
})
