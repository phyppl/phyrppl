#include "hip/hip_runtime.h"
/* 
 *  models/CombineDS.cuh
 *
 *  Copyright (C) 2020-2021 Viktor Senderov, Joey Öhman, David Broman
 * 
 *
 *  CombineDS diversification model supports conditionally simulates
 *  several different types of evolution:
 *
 *    - cladogenetic (ClaDS-like) small changes in diversification
 *      rates, ClaDS versions 0-2.
 *	      
 *    - anagenetic small changes (happening on a single lineage)
 *
 *    - rare large shits (both anagenetic and cladogenetic)
 *
 *    - uncoupling of the turnover rate at the rare large shifts for
 *      ClaDS2
 *
 *
 *  This file needs to be included by a .cu file, containing the MAIN
 *  macro, needed global parameters, needed tuning parameters, and 
 *  the tree structure as a datatype.
 * 
 *  Needed global parameters:
 * 
 *    const floating_t k = 1;            // prior Γ-shape for λ
 *    const floating_t theta = 1;        // prior Γ-scale for λ
 *
 *    const floating_t kNu = 1;          // prior Γ-shape for ν
 *    const floating_t thetaNu = 0.5;    // prior Γ-shape for ν
 *
 *    const floating_t a_epsilon = 1;    // prior β-shape 1 for p_ε
 *    const floating_t b_epsilon = 100;  // prior β-shape 2 for p_ε
 *
 *    const floating_t m0 = 0;   // Hyper-param of prior for α and σ
 *    const floating_t v = 1;    // Hyper-param of prior for α and σ
 *    const floating_t a = 1.0;  // Hyper-param of prior for α and σ
 *    const floating_t b = 0.2;  // Hyper-param of prior for α and σ
 * 
 *  Needed tuning parameters:
 *
 *    #define M 20              // Number of subsamples to draw
 *    #define RARE_SHIFT false  // Activate rare shifts
 *    #define CLADS true        // Cladogenetic changes
 *    #define ANADS true        // Anagenetic changes
 *    #define UNCOUPLE true     // Uncouples turnover rate at rare shifts
 *    #define CLADS1 false      // ClaDS version: 0, 1, or 2, TODO 0
 *
 *  Tree selection, 3 steps:
 *
 *    #include "trees/cetaceans.cuh"       // (1)
 *    typedef cetaceans_87_tree_t tree_t;  // (2)
 *    const floating_t rhoConst = 1.00;    // (3) sampling rate
 *
 *  models/CombineDS.cuh defines the following BBLOCKS that can be included
 *  in the MAIN macro:
 *
 *    - simCombinedDS         (required)
 *
 *    - simTree               (required)
 *
 *    - conditionOnDetection  (optional, corrects for survivorship bias)
 *
 *    - sampleFinalLambda     (optional, samples the global parameters,
 *                             which have been delayed)
 *
 *    - saveResults           (optional callback, needs to be used in 
 *                             conjunction with sampleFinalLambda)
 */

/* Preamble */
#include <iostream>
#include <cstring>
#include <cassert>
#include <string>
#include <fstream>
#include <algorithm>
#include <random>

#include "inference/smc/smc.cuh"
#include "trees/tree_utils.cuh"
#include "utils/math.cuh"
#include "utils/stack.cuh"
#include "dists/delayed.cuh"

#include "trees/cetaceans.cuh"
#include "trees/default_trees.cuh"




/////////////////////////////////////////////////////////////////////////////

#define STEPS 4


/* Program state */
struct progState_t {
  floating_t f = 0;
  floating_t f_1_2 = 0;
};


#define NUM_BBLOCKS 1

INIT_MODEL(progState_t)

/*
 * simCombineDS - required BBLOCK
 */
BBLOCK(test,
{
  normalInverseGamma_t alpha_sigma = normalInverseGamma_t(0, 1.0, 3.0, 1.0);
  floating_t f =  SAMPLE(sample_NormalInverseGammaNormal, alpha_sigma);
  PSTATE.f = f;

  floating_t factorSum = 0;
  normalInverseGamma_t alpha_sigma_nu = normalInverseGamma_t(0, 2*2, 3.0, 0.25);
  for (int i = 0; i < STEPS; i++) {
    factorSum += SAMPLE(sample_NormalInverseGammaNormal, alpha_sigma_nu);
  }

  PSTATE.f_1_2 = factorSum;
  NEXT = NULL;
})

CALLBACK(printResults, {
  floating_t meanF = 0;
  floating_t varF = 0;
  floating_t meanF1_2 = 0;
  floating_t varF1_2 = 0;
  for (int i = 0; i < N; i++) {
    meanF += PSTATES[i].f;
    meanF1_2 += PSTATES[i].f_1_2;
  }
  meanF = meanF / N; meanF1_2 = meanF1_2 / N;
  for (int i = 0; i < N; i++ ) {
    varF += pow(meanF - PSTATES[i].f, 2.0);
    varF1_2 += pow(meanF1_2 - PSTATES[i].f_1_2, 2.0);
  }
  varF = varF / (N - 1); varF1_2 = varF1_2 / (N - 1);
  printf("meanF\t\tvarF1\t\tmeanF1_2\t\tvarF1_2\n");
  printf("%f\t%f\t%f\t%f\n", meanF, varF, meanF1_2, varF1_2);
})

MAIN({
    FIRST_BBLOCK(test)
    SMC(printResults)
    //SMC(NULL)
})
